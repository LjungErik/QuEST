#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

#include "compression.h"

bool rawDataBlock_is_current_block(RawDataBlock* block, long long int block_idx) {
    return block->used && 
           block->mem_block_index == block_idx;
}

qreal rawDataBlock_get_value(RawDataBlock* block, long long int index) {
    if (!block->used) {
        printf("Error, cannot fetch data, block not used. Trying internal index: %lli\n", index);
        return 0.0;
    }

    if (block->n_values <= index) {
        printf("Error, cannot fetch index, index out of range");
        return 0.0;
    }

    return block->data[index];
}

bool rawDataBlock_set_value(RawDataBlock* block, long long int index, qreal value) {
    if (!block->used) {
        printf("Error, cannot set data, block not used. Trying interal index: %lli\n", index);
        return false;
    }

    if (block->n_values <= index) {
        printf("Error, cannot fetch index, index out of range");
        return false;
    }

    block->data[index] = value;
    return true;
}

size_t compression_maxSize(CompressionImp *imp) {
    return imp->max_size(imp->config);
}

void compression_compress(CompressionImp *imp,  CompressedBlock *out_block, RawDataBlock *in_block) {
    imp->compress(imp->config, out_block, in_block);
}

void compression_decompress(CompressionImp *imp, CompressedBlock *in_block, RawDataBlock *out_block) {
    imp->decompress(imp->config, in_block, out_block);
}

CompressedMemory* compressedMemory_allocate(CompressionConfig conf) {
    CompressedMemory *mem = NULL;
    hipMalloc(&mem, sizeof(CompressedMemory));

    mem->imp = conf.imp;
    mem->n_blocks = conf.n_blocks;
    mem->values_per_block = conf.values_per_block;

    size_t max_n = compression_maxSize(&mem->imp);

    /* Allocate the number of blocks */
    /* For each block allocate the memory for max_n */
    printf("Creating %li blocks for memory, Max block size %li, total Max size: %lli\n", 
                mem->n_blocks, max_n, (long long int) (mem->n_blocks * max_n));

    hipMalloc(&mem->blocks, (mem->n_blocks * sizeof(CompressedBlock)));
    for (int i = 0; i < mem->n_blocks; i++) {
        mem->blocks[i].n_values = mem->values_per_block;
        mem->blocks[i].max_size = max_n;
        mem->blocks[i].size = 0;
        // if dynamic allocation
        if (conf.use_dynamic_allocation) {
            mem->blocks[i].data = NULL;
        } else {
            hipMalloc(&(mem->blocks[i].data), max_n*sizeof(char));
        }
    }

    return mem;
}

void compressedMemory_destroy(CompressedMemory *mem) {
    for (int i = 0; i < mem->n_blocks; i++) {
        if (mem->blocks[i].data != NULL) {
            hipFree(mem->blocks[i].data);
        }
    }
    hipFree(mem->blocks);
    hipFree(mem);
}

void compressedMemory_save(CompressedMemory *mem, RawDataBlock* block) {
    if (!block->used) {
        printf("Error, cannot save uncompressed data, block contains no data.\n");
        return;
    }

    size_t index = block->mem_block_index;
    //printf("Compress block: %li\n", index);
    compression_compress(&mem->imp, &mem->blocks[index], block);

    block->used = false;
    block->n_values = 0;
}


void compressedMemory_load(CompressedMemory *mem, size_t index, RawDataBlock* block) {
    if (block->used) {
        compression_compress(&mem->imp, &mem->blocks[index], block);
    }

    compression_decompress(&mem->imp, &mem->blocks[index], block);

    block->mem_block_index = index;
    block->used = true;
}

qreal compressedMemory_get_value(CompressedMemory *mem, RawDataBlock *block, long long int index) {
    long long int block_idx = (index / mem->values_per_block);

    if (block_idx >= mem->n_blocks) {
        printf("Invalid block index, block index out of range, block index: %lli\n", block_idx);
        return 0.0;
    }

    // Calculate local index inside block (index -> internal block index, range. 0-1023) 
    long long int internal_idx = index - (block_idx * mem->values_per_block);
    
    // Check if RawDataBlock is the correct block index
    if (!rawDataBlock_is_current_block(block, block_idx)) {
        // Decompress the specific block
        compressedMemory_load(mem, block_idx, block);
    }  
        
    // Get specific data for interal index in the uncompressed raw data block
    return rawDataBlock_get_value(block, internal_idx);
}

void compressedMemory_set_value(CompressedMemory *mem, RawDataBlock *block, long long int index, qreal value) {
    long long int block_idx = (index / mem->values_per_block);

    if (block_idx >= mem->n_blocks) {
        printf("Invalid block index, block index out of range, block index: %lli\n", block_idx);
        return;
    }

    // Calculate local index inside block (index -> internal block index, range. 0-1023) 
    long long int internal_idx = index - (block_idx * mem->values_per_block);
    
    //printf("Setting block: %lli, internal index: %lli, value: %f\n", block_idx, internal_idx, value);

    // Check if RawDataBlock is the correct block index
    if (!rawDataBlock_is_current_block(block, block_idx)) {
        compressedMemory_load(mem, block_idx, block);
    }  

    rawDataBlock_set_value(block, internal_idx, value);
}

RawDataBlock* rawDataBlock_allocate(CompressionConfig conf) {
    RawDataBlock* block = NULL;
    hipMalloc(&block, sizeof(RawDataBlock));
    size_t data_size = (size_t) (conf.values_per_block * sizeof(*(block->data)));

    printf("Allocating Raw Data block with data size: %li\n", data_size);

    if (conf.use_dynamic_allocation) {
        size_t max_n = compression_maxSize(&conf.imp);
        hipMalloc(&(block->tmp_storage), (max_n * sizeof(char)));
        block->tmp_max_size = max_n;
    } else {
        block->tmp_storage = NULL;
        block->tmp_max_size = 0;
    }

    hipMalloc(&(block->data), data_size);
    block->size = data_size;
    block->n_values = 0;
    block->mem_block_index = 0;
    block->used = false;

    return block;
}

void rawDataBlock_destroy(RawDataBlock* block) {
    if (block->tmp_storage != NULL) {
        hipFree(block->tmp_storage);
    }

    hipFree(block->data);
    hipFree(block);
}

void rawDataBlock_dump_to_file(RawDataBlock *block, FILE *stream) {
    if (!block->used) {
        printf("Missing data, raw data block does not contain any data\n");
        return;
    }

    /* Fire and forget */
    fwrite(block->data, sizeof(*(block->data)), block->n_values, stream);
}

void compressedMemory_dump_memory_to_file(CompressedMemory *mem, RawDataBlock *block, FILE *stream) {
    /* if in use, save current state */
    if (block->used) {
        // Compress the existing block and save to memory
        compressedMemory_save(mem, block);
    }

    for(size_t i = 0; i < mem->n_blocks; i++) {
        block->used = false;
        compressedMemory_load(mem, i, block);
        rawDataBlock_dump_to_file(block, stream);
    }
}